#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <fstream>
#include <iomanip>

#include <stdio.h>
#include <fstream>
#include <iostream>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// The cuda kernel
__global__ void quamsim_kernel( const float *A, 
                                const float *B, 
                                float *C, 
                                int q_bit, 
                                int numElements) {

    int i = threadIdx.x + blockIdx.x * blockDim.x; 
    int j = i/q_bit;

    //A -> Quantum Gate
    //B -> Input Vector; C -> Output Vector
    //q_bit -> target q-bit; size -> total number of elements
    
    if((i<numElements)&&((i+q_bit)<numElements)&&(j%2==0)) {
        C[i]        = A[0]*B[i] + A[1]*B[i+q_bit];
        C[i+q_bit]  = A[2]*B[i] + A[3]*B[i+q_bit];
    }

    // q_bit = q_bit>>1;
    // if (i < numElements) {
    //     // Calculate the index of the adjacent element
    //     bool control_qubit_set = ((i >> q_bit) & 1) == 1;
    //     int j = (control_qubit_set) ? i ^ (1 << q_bit) : i; // Correct calculation  
    //     // // Apply the quantum gate operation
    //     if(j<numElements){
    //     if (control_qubit_set) {
    //         C[i] = A[0] * B[i] + A[1] * B[j]; // Apply with correct index swapping
    //         C[j] = A[2] * B[i] - A[3] * B[j]; // Correct signs for A[2] and A[3]
    //     } else {
    //         C[i] = A[2] * B[i] + A[3] * B[j];
    //         C[j] = A[0] * B[i] + A[1] * B[j];
    //     }
    //     }
    // }
}

int main(int argc, char *argv[]) {

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Read the inputs from command line
    if (argc != 2) {
        printf("Error: Please provide the input file\n");
        return EXIT_FAILURE;
    }

    const std::string file = argv[1];

    // Open the input file and read the data
    std::ifstream inputFile(file);
    if (!inputFile.is_open()) {
        printf("Error: Could not open the input file\n");
        return EXIT_FAILURE;
    }

    int count = 0;
    float temp;
    while (inputFile >> temp) {
        count++;
    }

    inputFile.close();

    int numElements = count - 5;
    int gate_size = 4*sizeof(float);
    float *A = (float *)malloc(gate_size);
    int arr_size = numElements*sizeof(float);
    float *B = (float *)malloc(arr_size);
    float *C = (float *)malloc(arr_size);
    int q_bit;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    inputFile.open(file);
    // Read the gate matrix elements
    for (int i = 0; i < 4; i++) {
        inputFile >> A[i];
    }
    // Read the state vector elements
    for (int i = 0; i < numElements; i++) {
        inputFile >> B[i]; 
    }
    // Read the gate target qubit index
    inputFile >> q_bit;
    q_bit = 1<<q_bit;
    inputFile.close();
    


    // Allocate/move data using hipMalloc and cudaMemCpy
    // float *d_A, *d_B, *d_C;
    // hipMalloc(&d_A, gate_size);
    // hipMalloc(&d_B, arr_size);
    // hipMalloc(&d_C, arr_size);
    // hipMemcpy(d_A, A, gate_size, hipMemcpyHostToDevice);
    // hipMemcpy(d_B, B, arr_size, hipMemcpyHostToDevice);

    // Allocate the device input vector A (2X2 Qunatum Gate)
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, gate_size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B (Qunatum State Vector)
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, arr_size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Allocate the device input vector C (Qunatum State Vector - output)
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, arr_size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, A, gate_size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, B, arr_size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    // Launch the kernel and take timestamps before and after
    hipEventRecord(start);
    quamsim_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, q_bit, numElements);
    hipEventRecord(stop);
    err = hipGetLastError();


    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(C, d_C, arr_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize(); // Wait for kernel to finish

    // // Copy the output back to host memory
    // hipMemcpy(C, d_C, arr_size, hipMemcpyDeviceToHost);
    // Print the output
    printf("Quantum Gate Matrix A:\n");
    for(int i=0; i<4; i++){
    printf("%.3f ", A[i]);
    if ((i+1) % 2 == 0) printf("\n");
    }

    printf("Input Vector B:\n");
    for(int i=0; i<numElements; i++){
    printf("%.3f \n", B[i]);
    }

    printf("q_bit = %d\n", q_bit);

    printf("Output Vector C:\n");
    for(int i=0; i<numElements; i++){
    printf("%.3f \n", C[i]);
    }


    // Extract the timing information
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f\n", milliseconds);
    // printf("%f\n", hipEventElapsedTime);


    // Clean up the memory
    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(A);
    free(B);
    free(C);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}




/**
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

**/
